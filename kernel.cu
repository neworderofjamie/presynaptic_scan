#include "hip/hip_runtime.h"
// Standard C++ includes
#include <algorithm>
#include <chrono>
#include <iostream>
#include <numeric>
#include <random>
#include <stdexcept>
#include <string>
#include <sstream>
#include <tuple>
#include <vector>

// Standard C includes
#include <cassert>
#include <cmath>

// CUDA includes
#include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <>

//------------------------------------------------------------------------
// Macros
//------------------------------------------------------------------------
#define NUM_POPULATIONS 50
#define SEED 124

#define CHECK_CUDA_ERRORS(call) {                                                                   \
    hipError_t error = call;                                                                       \
    if (error != hipSuccess) {                                                                     \
            std::ostringstream errorMessageStream;                                                  \
            errorMessageStream << "cuda error:" __FILE__ << ": " << __LINE__ << " ";                \
            errorMessageStream << hipGetErrorString(error) << "(" << error << ")" << std::endl;    \
            throw std::runtime_error(errorMessageStream.str());                                     \
        }                                                                                           \
    }


template<typename T>
using HostDeviceArray = std::pair < T*, T* > ;

struct MergedPresynapticUpdateGroup
{
    float *inSyn;
    unsigned int *srcSpkCnt;
    unsigned int *srcSpk;
    float *weight;
};

// Host globals
unsigned int mergedGroupStartID[NUM_POPULATIONS];
MergedPresynapticUpdateGroup mergedGroups[NUM_POPULATIONS];

// Device globals
__device__ unsigned int d_mergedGroupStartID[NUM_POPULATIONS];
__device__ __constant__  MergedPresynapticUpdateGroup d_mergedGroups[NUM_POPULATIONS];

__global__ void presynapticUpdateIdleThreads()
{
    const unsigned int id = threadIdx.x + (blockIdx.x * blockDim.x);

    unsigned int lo = 0;
    unsigned int hi = NUM_POPULATIONS;
    while(lo < hi) {
        const unsigned int mid = (lo + hi) / 2;
        if(id < d_mergedGroupStartID[mid]) {
            hi = mid;
        }
        else {
            lo = mid + 1;
        }
    }
    struct MergedPresynapticUpdateGroup *group = &d_mergedGroups[lo - 1]; 
    const unsigned int groupStartID = d_mergedGroupStartID[lo - 1];
    const unsigned int lid = id - groupStartID;

    if(lid < group->srcSpkCnt[0]) {
        const unsigned int preInd = group->srcSpk[lid];
        atomicAdd(&group->inSyn[preInd], group->weight[preInd]);
    }
}


//-----------------------------------------------------------------------------
// Host functions
//-----------------------------------------------------------------------------
template<typename T>
HostDeviceArray<T> allocateHostDevice(size_t count)
{
    T *array = nullptr;
    T *d_array = nullptr;
    CHECK_CUDA_ERRORS(hipHostMalloc(&array, count * sizeof(T)));
    CHECK_CUDA_ERRORS(hipMalloc(&d_array, count * sizeof(T)));

    return std::make_pair(array, d_array);
}
//-----------------------------------------------------------------------------
template<typename T>
void hostToDeviceCopy(HostDeviceArray<T> &array, size_t count, bool deleteHost=false)
{
    CHECK_CUDA_ERRORS(hipMemcpy(array.second, array.first, sizeof(T) * count, hipMemcpyHostToDevice));
    if (deleteHost) {
        CHECK_CUDA_ERRORS(hipHostFree(array.first));
        array.first = nullptr;
    }
}
//-----------------------------------------------------------------------------
template<typename T>
void deviceToHostCopy(HostDeviceArray<T> &array, size_t count)
{
    CHECK_CUDA_ERRORS(hipMemcpy(array.first, array.second, count * sizeof(T), hipMemcpyDeviceToHost));
}
//-----------------------------------------------------------------------------
int main(int argc, char *argv[])
{
    const unsigned int popSize = 5000;
    const unsigned int numSpikes = 500;
    const unsigned int blockSize = 32;
    const bool oracle = true;

    const unsigned int paddedPopSize = ((popSize + blockSize - 1) / blockSize) * blockSize;
    std::cout << "Padded pop size:" << paddedPopSize << std::endl;

    CHECK_CUDA_ERRORS(hipSetDevice(0));

    hipEvent_t updateStart;
    hipEvent_t updateEnd;
    CHECK_CUDA_ERRORS(hipEventCreate(&updateStart));
    CHECK_CUDA_ERRORS(hipEventCreate(&updateEnd));

    std::mt19937 rng;
    std::uniform_int_distribution<unsigned int> spikeDist(0, popSize - 1);
    std::normal_distribution<float> weightDist(0.0f, 0.25f);

    HostDeviceArray<float> inSyn[NUM_POPULATIONS];
    std::vector<float> correctInSyn[NUM_POPULATIONS];

    unsigned int startThread = 0;
    for(unsigned int i = 0; i < NUM_POPULATIONS; i++) {
        // Resize and zero correct insyn vector
        correctInSyn[i].resize(popSize, 0.0f);

        // Allocate memory
        inSyn[i] = allocateHostDevice<float>(popSize);
        auto srcSpkCnt = allocateHostDevice<unsigned int>(1);
        auto srcSpk = allocateHostDevice<unsigned int>(popSize);
        auto weight = allocateHostDevice<float>(popSize);

        // Zero inSyn
        std::fill_n(&inSyn[i].first[0], popSize, 0.0f);
        

        // Generate random spikes
        srcSpkCnt.first[0] = numSpikes;
        std::generate_n(&srcSpk.first[0], numSpikes, [&rng, &spikeDist]() { return spikeDist(rng); });

        // Generate weights
        std::generate_n(&weight.first[0], popSize, [&rng, &weightDist]() { return weightDist(rng); });

        // Calculate correct output
        for(unsigned int j = 0; j < numSpikes; j++) {
            const unsigned int ind = srcSpk.first[j];
            correctInSyn[i][ind] += weight.first[ind];
        }

        // Upload
        hostToDeviceCopy(inSyn[i], popSize);
        hostToDeviceCopy(srcSpkCnt, 1, true);
        hostToDeviceCopy(srcSpk, popSize, true);
        hostToDeviceCopy(weight, popSize, true);

        // Build struct with device pointers
        mergedGroups[i].inSyn = inSyn[i].second;
        mergedGroups[i].srcSpk = srcSpk.second;
        mergedGroups[i].srcSpkCnt = srcSpkCnt.second;
        mergedGroups[i].weight = weight.second;

        // Calculate static start ID
        mergedGroupStartID[i] = oracle ? startThread : (i * paddedPopSize);
        
        // Sum padded spikes
        startThread += ((numSpikes + blockSize - 1) / blockSize) * blockSize;        
    }

    // Copy merged group structures to symbols
    CHECK_CUDA_ERRORS(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_mergedGroups), &mergedGroups[0], sizeof(MergedPresynapticUpdateGroup) * NUM_POPULATIONS));
    CHECK_CUDA_ERRORS(hipMemcpyToSymbolAsync(HIP_SYMBOL(d_mergedGroupStartID), &mergedGroupStartID[0], sizeof(unsigned int) * NUM_POPULATIONS));

    {
        const unsigned int numBlocks = oracle ? (startThread / blockSize) : ((paddedPopSize / blockSize) * NUM_POPULATIONS);
        dim3 threads(blockSize, 1);
        dim3 grid(numBlocks, 1);

        CHECK_CUDA_ERRORS(hipEventRecord(updateStart));
        presynapticUpdateIdleThreads<<<grid, threads>>>();
        CHECK_CUDA_ERRORS(hipEventRecord(updateEnd));
        CHECK_CUDA_ERRORS(hipEventSynchronize(updateEnd));
        float time;
        CHECK_CUDA_ERRORS(hipEventElapsedTime(&time, updateStart, updateEnd));
        std::cout << "Idle threads:" << time << std::endl;
    }

    for(unsigned int i = 0; i < NUM_POPULATIONS; i++) {
        deviceToHostCopy(inSyn[i], popSize);

        for(unsigned int j = 0; j < popSize; j++) {
            if(std::fabs(inSyn[i].first[j] - correctInSyn[i][j]) > 0.0001f) {
                std::cerr << "ERROR" << std::endl;
            }
        }
    }

    return EXIT_SUCCESS;
}
